#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
  // Print a message only for the last thread (1023) of each block
  // Thread id in a block
  if (threadIdx.x == 1023)
  {
     printf("Hello world from thread %d, block %d\n", threadIdx.x, blockIdx.x);
     printf("Name: Diego Alonzo, Carne: 20172\n");
  }
}

int main()
{
  // 2 blocks of 1024 threads each
  hello<<<2,1024>>>();
  hipDeviceSynchronize(); //deprecated
  return 0;
}
