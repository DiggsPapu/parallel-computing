#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0); // Assuming device 0

    printf("Warp size: %d\n", props.warpSize);
    printf("Max threads per block: %d\n", props.maxThreadsPerBlock);
    printf("Max block dimensions (x, y, z): (%d, %d, %d)\n",
            props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2]);
    printf("Max grid dimensions (x, y, z): (%d, %d, %d)\n",
            props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);

    // Maximum size per grid dimension (x, y, z)
    int maxGridSizeX = props.maxGridSize[0];
    int maxGridSizeY = props.maxGridSize[1];
    int maxGridSizeZ = props.maxGridSize[2];

    // Maximum size per block dimension (x, y, z)
    int maxBlockSizeX = props.maxThreadsDim[0];
    int maxBlockSizeY = props.maxThreadsDim[1];
    int maxBlockSizeZ = props.maxThreadsDim[2];

    printf("Max grid size X: %d\n", maxGridSizeX);
    printf("Max grid size Y: %d\n", maxGridSizeY);
    printf("Max grid size Z: %d\n", maxGridSizeZ);

    printf("Max block size X: %d\n", maxBlockSizeX);
    printf("Max block size Y: %d\n", maxBlockSizeY);
    printf("Max block size Z: %d\n", maxBlockSizeZ);

    return 0;
}